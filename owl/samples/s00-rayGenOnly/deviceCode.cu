// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  if (pixelID == owl::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program%s\n",
           OWL_TERMINAL_CYAN,
           OWL_TERMINAL_DEFAULT);
  }
  if (pixelID.x >= self.fbSize.x) return;
  if (pixelID.y >= self.fbSize.y) return;

  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  const vec3f color = (pattern&1) ? self.color1 : self.color0;

  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
  self.fbPtr[fbOfs]
    = owl::make_rgba(color);
}

