#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

OPTIX_INTERSECT_PROGRAM(Sphere)()
{
  const SphereGeomData &self = owl::getProgramData<SphereGeomData>();
  // get index of primitive we are to intersect (for this example,
  // this will always be 0, because we have N differnent *geoms* with
  // one prim each.
  int primID = optixGetPrimitiveIndex();
  
  const vec3f org  = optixGetWorldRayOrigin();
  const vec3f dir  = optixGetWorldRayDirection();
  float hit_t      = optixGetRayTmax();
  const float tmin = optixGetRayTmin();

  const vec3f oc = org - self.center;
  const float a = dot(dir,dir);
  const float b = dot(oc, dir);
  const float c = dot(oc, oc) - self.radius * self.radius;
  const float discriminant = b * b - a * c;
  
  if (discriminant < 0.f) return;

  {
    float temp = (-b - sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
      
  {
    float temp = (-b + sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
  if (hit_t < optixGetRayTmax()) {
    optixReportIntersection(hit_t, 0);
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(Sphere)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const SphereGeomData &self = owl::getProgramData<SphereGeomData>();
  
  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f Ng     = normalize(hit_P-self.center);

  prd = (.2f + .8f*fabs(dot(dir,Ng)))*self.color;
}

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = (pattern&1) ? self.color1 : self.color0;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  
  if (pixelID.x >= self.fbSize.x) return;
  if (pixelID.y >= self.fbSize.y) return;

  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;
  ray.origin    
    = self.camera.pos;
  ray.direction 
    = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);

  vec3f color;
  owl::trace(/*accel to trace against*/self.world,
             /*the ray to trace*/ ray,
             /*numRayTypes*/1,
             /*prd*/color);
    
  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
  self.fbPtr[fbOfs]
    = owl::make_rgba(color);
}


