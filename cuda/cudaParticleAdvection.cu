#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 The Collaborators                                    //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //


#include "common.h"
#include "DeviceTetMesh.cuh"
#include "HostTetMesh.h"

#include "query/ConvexQuery.h"
#include "query/RTQuery.h"
#include "optix/OptixQuery.h"



//Switch between RTX and ConvexPoly
//#define ConvexPoly
#define RTX

namespace advect {

	//Input parameters
	int numParticles = 1e3;
	int testTetMeshGridSize = 45;
	//int numSteps = 550001;
	//int numSteps = 13251;
	int numSteps = 1e5;
	//double dt = 1.5e-3; //Microfludics Level 1
	double dt = 3.5e-3; //Microfludics Level 2
	//double dt = 5.0e-3; //Microfludics Level 3
	//double dt = 6.0e-3; //Microfludics Level 4

	//double dt = 1e-2;
	//double dt = 1e-3;

	bool fixTimestep = true;

	double tol = 1e-5;
	double SeedingBox[6] = { 0.0 + tol, 0.0 + tol, 0.0 + tol,
							 1.0 - tol, 1.0 - tol, 1.0 - tol };

	//Physics controller
	bool usingAdvection = true;
	bool usingBrownianMotion = true;
	bool reflectWall = true;
	bool floatErrorCorrection = true;

	//IO container
	bool usingSeedingBox = false;
	bool saveStreamlinetoFile = false;
	int saveInterval = numSteps;
	//int saveInterval = 50000;

  extern "C" int main(int ac, char **av)
  {
    //cudaTimer timer;
	CPUTimer timer;

    timer.start();

	std::string vert_filename, tet_filename, velocity_vert_filename,velocity_tet_filename;
	std::string seeding_pts_filename;

	seeding_pts_filename = "SeedingPts.dat";
	seeding_pts_filename = "";
	vert_filename = "verts.dat";
	tet_filename = "cells.dat";
	velocity_tet_filename = "solutions_cell.dat";


	std::string objTrajectoryFileName;
	std::string vtkStreamlineFileName = "Streamline.vtk";
    for (int i=1;i<ac;i++) {
      const std::string arg = av[i];
      if (arg == "--num-particles")
        numParticles = std::atoi(av[++i]);
      else if (arg == "--num-steps")
        numSteps = std::atoi(av[++i]);
	  else if (arg == "--input_mesh") {
		  vert_filename = av[++i];
		  tet_filename = av[++i];
	  }
	  else if (arg == "--input_vertex_velocity_field") 
		  velocity_vert_filename= av[++i];
	  else if (arg == "--input_tet_velocity_field")
		  velocity_tet_filename = av[++i];
	  else if (arg == "--test-grid-size")
		  testTetMeshGridSize = std::atoi(av[++i]);
	  else if (arg == "-dt")
		  dt = std::atof(av[++i]);
	  else if (arg == "--seeding-box") {
		  for (int si = 0; si < 6; ++si)
			  SeedingBox[si] = std::atof(av[++i]);
		  usingSeedingBox = true;
	  }
	  else if (arg == "--save-streamline-to-obj") {
		  objTrajectoryFileName = av[++i];
		  saveStreamlinetoFile = true;
	  }
	  else if (arg == "--save-streamline-to-vtk") {
		  vtkStreamlineFileName = av[++i];
		  saveStreamlinetoFile = true;
	  }
      else
        throw std::runtime_error("unknown cmdline argument '"+arg+"'");
    }

    // ------------------------------------------------------------------
    // create a host-side model
    // ------------------------------------------------------------------
	std::string VelocityInterpMethod = "VertexVelocity";
	HostTetMesh hostTetMesh;
	if (velocity_vert_filename.size() > 0) {
		hostTetMesh = HostTetMesh::readDataSet(vert_filename, tet_filename, velocity_vert_filename);
		VelocityInterpMethod = "VertexVelocity";
		printf("#adv: load vertex velocity field from file %s", velocity_vert_filename.c_str());
	}
	else if (velocity_tet_filename.size() > 0) {
		hostTetMesh = HostTetMesh::readDataSet(vert_filename, tet_filename, "", velocity_tet_filename);
		VelocityInterpMethod = "TetVelocity";
		printf("#adv: load tet velocity field from file %s", velocity_vert_filename.c_str());
	}
	else {
		hostTetMesh = HostTetMesh::createBoxMesh(testTetMeshGridSize, testTetMeshGridSize, testTetMeshGridSize);
		VelocityInterpMethod = "VertexVelocity";
		printf("#adv: using synthetic velocity field\n");
	}
	//Get the boundary mesh representation
	HostTetMesh hostBoundaryMesh = hostTetMesh.getBoundaryMesh();

	std::cout << OWL_TERMINAL_YELLOW
		<< "#adv: mesh mem: "
		<< prettyNumber(hostTetMesh.bytes())
		<< OWL_TERMINAL_DEFAULT
		<< std::endl;

    // ------------------------------------------------------------------
    // build the query accelerator first, before the cuda kernels
    // allocate their memory.
    // ------------------------------------------------------------------
    OptixQuery tetQueryAccelerator((double3 *) hostTetMesh.positions.data(),
                                      hostTetMesh.positions.size(),
                                      (int4 *) hostTetMesh.indices.data(),
                                      hostTetMesh.indices.size());

    // by now optix should have built all its data,and released
    // whatever temp memory it has used.
	OptixQuery triQueryAccelerator((double3*)hostBoundaryMesh.positions.data(),
									hostBoundaryMesh.positions.size(),
									(int4*)hostBoundaryMesh.indices.data(),
									hostBoundaryMesh.indices.size(), true);

    // ------------------------------------------------------------------
    // upload our own cuda data
    // ------------------------------------------------------------------
    DeviceTetMesh devMesh;
    devMesh.upload(hostTetMesh);

	DeviceBdMesh devBdMesh;
	devBdMesh.upload(hostBoundaryMesh);

    // ------------------------------------------------------------------
    // now run sample advection...
    // ------------------------------------------------------------------
	if(!fixTimestep)
	double dt=cudaEvalTimestep(hostTetMesh.indices.size(),
		devMesh.d_indices,
		devMesh.d_positions,
		devMesh.d_velocities,
		VelocityInterpMethod);

    // alloc particles and its properties
	
	// Cast simple double4 particles into OptixTetquery type
	Particle* d_particles = nullptr;//x,y,z,statusID
	if (seeding_pts_filename.size() > 0) 
		numParticles = loadNumParticles(seeding_pts_filename);
	cudaCheck(hipMalloc(&d_particles, numParticles * sizeof(Particle)));
	int* d_particles_tetIDs = nullptr;
	cudaCheck(hipMalloc(&d_particles_tetIDs, numParticles * sizeof(int)));

	int* d_particles_triIDs = nullptr;
	cudaCheck(hipMalloc(&d_particles_triIDs, numParticles * sizeof(int)));

	int* d_particles_ConvextetIDs = nullptr;
	cudaCheck(hipMalloc(&d_particles_ConvextetIDs, numParticles * sizeof(int)));
	cudaCheck(hipMemset(d_particles_ConvextetIDs, -1, numParticles * sizeof(int)));

	vec4d* d_particle_disps = nullptr;//x,y,z,tetID_last
	cudaCheck(hipMalloc(&d_particle_disps, numParticles * sizeof(vec4d)));
	cudaCheck(hipMemset(d_particle_disps, 0.0, numParticles * sizeof(vec4d)));

	vec4d* d_particle_vels =nullptr;//vx,vy,vz
	cudaCheck(hipMalloc(&d_particle_vels, numParticles * sizeof(vec4d)));
	cudaCheck(hipMemset(d_particle_vels, 0.0, numParticles * sizeof(vec4d)));

	hiprandState_t* rand_states = nullptr;;
	cudaCheck(hipMalloc(&rand_states, numParticles * sizeof(hiprandState_t)));
	initRandomGenerator(numParticles, rand_states);

	size_t bytes = numParticles * sizeof(Particle)
		+ numParticles * sizeof(int)
		+ numParticles * sizeof(vec4d)
		+ numParticles * sizeof(vec4d)
		+ numParticles * sizeof(hiprandState_t);
	std::cout << OWL_TERMINAL_YELLOW
		<< "#adv: particle mem: "
		<< prettyNumber(bytes)
		<< OWL_TERMINAL_DEFAULT
		<< std::endl;

	// Create streamlines object
	std::vector<std::vector<vec3f>> trajectories;


	usingSeedingBox = true;
	//double seedBox[6] = { -0.05+tol,0.0+tol,2.5+tol, 0.05-tol,0.1-tol,2.55-tol };
	//double seedBox[6] = { -0.05 + tol,0.0 + tol,2.2165 + tol, 0.05 - tol,0.1 - tol,2.7835 - tol }; //Square Duct
	//double seedBox[6] = { 73.9 + tol,-0.4 + tol,-655.95 + tol, 77.7 - tol,0.0 - tol,-655.45 - tol }; //Microfludics
	double seedBox[6] = { 6.5 + tol,6.5 + tol,-20 + tol, 91.5 - tol, 91.5 - tol,-16 - tol };//Sphere packing
	//double seedBox[6] = { 167.25 + tol,178.9 + tol,-63.4 + tol, 176.75 - tol, 188.7 - tol,-58.6 - tol };//Human lung
	std::copy(seedBox, seedBox+6, SeedingBox);


    // initialize with random particles
	if (seeding_pts_filename.size() == 0) {
		box3d initBox;
		if (usingSeedingBox) {
			initBox.extend(vec3d(SeedingBox[0], SeedingBox[1], SeedingBox[2]));
			initBox.extend(vec3d(SeedingBox[3], SeedingBox[4], SeedingBox[5]));
		}
		else {
			initBox = hostTetMesh.worldBounds;
		}
		//std::cout << "Particle seeding bounding box = " << initBox.lower << " " << initBox.upper << std::endl;
		printf("Particle Bounding Box (%lf,%lf,%lf)-(%lf,%lf,%lf)\n",
			initBox.lower.x, initBox.lower.y, initBox.lower.z,
			initBox.size().x, initBox.size().y, initBox.size().z);
		cudaInitParticles(d_particles, numParticles, initBox);
	}
	else 
		cudaInitParticles(d_particles, numParticles, seeding_pts_filename);

    cudaCheck(hipDeviceSynchronize());
    
    printf("Init RunTime=%lf  ms\n", timer.stop());

	//Init initial state (pos,velocity,tetID)
#ifdef  ConvexPoly
	RTQuery(tetQueryAccelerator, devMesh,
		d_particles, d_particles_ConvextetIDs, numParticles);
#else
	RTQuery(tetQueryAccelerator, devMesh,
		d_particles, d_particles_tetIDs, numParticles);
#endif


#ifndef  ConvexPoly
	cudaReportParticles(numParticles, d_particles_tetIDs);
#else
	cudaReportParticles(numParticles, d_particles_ConvextetIDs);
#endif

	if (usingAdvection) {
	
	cudaAdvect(d_particles,
#ifndef  ConvexPoly
		d_particles_tetIDs,
#else
		d_particles_ConvextetIDs,
#endif
		d_particle_vels,
		d_particle_disps,
		dt,
		numParticles,
		devMesh.d_indices,
		devMesh.d_positions,
		devMesh.d_velocities,
		VelocityInterpMethod);
	/*
	cudaTubeAdvect(d_particles, d_particles_tetIDs,
		           d_particle_vels, d_particle_disps, dt, numParticles);
	*/
	}
	
	writeParticles2VTU(0, d_particles, d_particle_vels, d_particles_tetIDs, numParticles,
		d_particles_ConvextetIDs);

#ifndef  ConvexPoly
	//testRT(tetQueryAccelerator, devMesh);
#else
	//testNStracing(tetQueryAccelerator, devMesh);
#endif

	system("pause");

	//VelocityInterpMethod = "ConstantVelocity";
	//VelocityInterpMethod = "VertexVelocity";

	//cudaTimer timer_loop;
	CPUTimer timer_loop;

	double advectionTime = 0.0;
	double diffusionTime = 0.0;
	double queryTime = 0.0;
	double reflectTime = 0.0;
	double moveTime = 0.0;
	double IOTime = 0.0;

	// and iterate
    timer.start();
    for (int i=1;i<=numSteps;i++) {
		if (i % 100 == 0) printf("------------Step %d-------------\n",i);
		// first, compute each particle's current tet for velocity interpolation
		//tetQueryAccelerator.query_sync(d_particles, d_particles_tetIDs, numParticles);

		// ... compute advection
		timer_loop.start();
		if (usingAdvection) {

			cudaAdvect(d_particles,
#ifndef  ConvexPoly
				d_particles_tetIDs,
#else
				d_particles_ConvextetIDs,
#endif
				d_particle_vels,
				d_particle_disps,
				dt,
				numParticles,
				devMesh.d_indices,
				devMesh.d_positions,
				devMesh.d_velocities,
				VelocityInterpMethod);
			/*
			cudaTubeAdvect(d_particles, d_particles_tetIDs,
						   d_particle_vels, d_particle_disps, dt, numParticles);
			*/
		}
		advectionTime+= timer_loop.stop();

		// ... compute random Brownian motion
		timer_loop.start();
		if(usingBrownianMotion)
		cudaBrownianMotion(d_particles, 
			d_particle_disps,
			rand_states,
			dt, 
			numParticles);
		diffusionTime += timer_loop.stop();


#ifndef  ConvexPoly

		timer_loop.start();
		///*
		RTQuery(devMesh,d_particles,d_particle_disps,d_particles_tetIDs,numParticles);

		//RTQuery(tetQueryAccelerator, devMesh,d_particles,d_particle_disps,d_particles_tetIDs,numParticles);
		queryTime += timer_loop.stop();


		timer_loop.start();
		if (reflectWall)
		RTWallReflect(
			devMesh,
			d_particles_tetIDs,
			d_particles,
			d_particle_disps,
			d_particle_vels,
			numParticles);
		//*/
		reflectTime += timer_loop.stop();
#else  
		timer_loop.start();
		// ... Convex Query particle tet location
		convexTetQuery(devMesh, 
			d_particles,
			d_particle_disps, 
			d_particles_ConvextetIDs, 
			numParticles);
		queryTime += timer_loop.stop();

		timer_loop.start();
		// ... compute wall reflection
		if (reflectWall)
		convexWallReflect(devMesh, 
			d_particles_ConvextetIDs,
			d_particles, 
			d_particle_vels,
			d_particle_disps, 
			numParticles);
		reflectTime += timer_loop.stop();
		//debug
		//RTQuery(devMesh,d_particles,d_particle_disps,d_particles_tetIDs,numParticles);
		//cudaReportParticles(numParticles, d_particles_tetIDs);
#endif



		timer_loop.start();
#ifndef  ConvexPoly
		// ... Move particles
		cudaMoveParticles(d_particles, d_particle_disps,
			numParticles, d_particles_tetIDs);
#else
		// ... Move particles
		cudaMoveParticles(d_particles, d_particle_disps,
			numParticles, d_particles_ConvextetIDs);
#endif
		moveTime += timer_loop.stop();


#ifndef  ConvexPoly
//		cudaReportParticles(numParticles, d_particles_tetIDs);
#else
//		cudaReportParticles(numParticles, d_particles_ConvextetIDs);
#endif

		timer_loop.start();
		if (saveStreamlinetoFile)
			if ((i % (saveInterval * 1)) == 0 || i == numSteps)
				addToTrajectories(d_particles, numParticles, trajectories);

		if ((i % saveInterval) == 0 || i == numSteps)
			writeParticles2VTU(i + 1, d_particles, d_particle_vels, d_particles_tetIDs, numParticles,
				d_particles_ConvextetIDs);
		IOTime += timer_loop.stop();

		if (i % 100 == 0) printf("------------End Step %d-------------\n\n", i);

    }
    std::cout << "#adv: advection steps = " << numSteps << std::endl;
    std::cout << "done ... ignoring proper cleanup for now" << std::endl;

#ifndef  ConvexPoly
	cudaReportParticles(numParticles, d_particles_tetIDs);
#else
	cudaReportParticles(numParticles, d_particles_ConvextetIDs);
#endif



	double runtime = timer.stop();
    printf("#adv: Simulation RunTime=%f ms\n", runtime);
	printf("#adv: Simulation Performance=%f steps/secs\n", numSteps/runtime*1000);
	
	double totalTime = advectionTime + diffusionTime + queryTime + reflectTime + moveTime + IOTime;
	printf("\tItem\ttime(s)\tfraction(%%)\n");
	printf("\tAdv\t%.2f\t\%.2f\n", advectionTime/1000, advectionTime / totalTime * 100);
	printf("\tDfs\t%.2f\t\%.2f\n", diffusionTime / 1000, diffusionTime / totalTime * 100);
	printf("\tQry\t%.2f\t\%.2f\n", queryTime / 1000, queryTime / totalTime * 100);
	printf("\tRft\t%.2f\t\%.2f\n", reflectTime / 1000, reflectTime / totalTime * 100);
	printf("\tMov\t%.2f\t\%.2f\n", moveTime / 1000, moveTime / totalTime * 100);
	printf("\tIO\t%.2f\t\%.2f\n", IOTime / 1000, IOTime / totalTime * 100);
	printf("\tTotal Time = %.2f ms\n", totalTime);
	printf("\tPerformance = %f steps/secs\n", numSteps / totalTime * 1000);

	if (saveStreamlinetoFile) {
		if (objTrajectoryFileName.size() > 0)
			saveTrajectories(objTrajectoryFileName, trajectories);
		if (vtkStreamlineFileName.size() > 0)
			writeStreamline2VTK(vtkStreamlineFileName, trajectories);
	}
	
	

    return 0;
  }

}
